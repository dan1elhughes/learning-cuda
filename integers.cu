
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c) {
	*c = *a + *b;
}

int main(void) {
    int a, b, c; // host copies of a, b, c
    int *gpu_a, *gpu_b, *gpu_c; // device copies of a, b, c
    int size = sizeof(int);

    // Allocate space for device copies of a, b, c
    hipMalloc((void **) &gpu_a, size);
    hipMalloc((void **) &gpu_b, size);
    hipMalloc((void **) &gpu_c, size);

    // Setup input values
    a = 2;
    b = 7;

    // Copy inputs to device
    hipMemcpy(gpu_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(gpu_b, &b, size, hipMemcpyHostToDevice);

    // Launch add() kernel on GPU
    add <<<1, 1>>> (gpu_a, gpu_b, gpu_c);

    // Copy result back to host
    hipMemcpy(&c, gpu_c, size, hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(gpu_a);
    hipFree(gpu_b);
    hipFree(gpu_c);
    printf("%i", c);
    return 0;
}
