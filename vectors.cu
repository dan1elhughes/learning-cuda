
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCKS 4
#define THREADSPERBLOCK 4
#define VECSIZE 16

__global__ void vector_add(int *a, int *b, int *c) {
	int i = (blockIdx.x * THREADSPERBLOCK) + threadIdx.x;
	c[i] = a[i] + b[i];
}

void random_ints(int* a, int n) {
	int i;
	for (i = 0; i < n; ++i) {
		a[i] = rand() % n;
	}
}

void show_vector(int* vec, int size) {
	for(int i = 0; i < size; i++) {
		printf("%d\t", vec[i]);
	}
	printf("\n");
}

int main(void) {
	int vector1[VECSIZE];
	int vector2[VECSIZE];
	int vector_out[VECSIZE];

	int *gpu_vector1, *gpu_vector2, *gpu_vector_out;

	int size = sizeof(int) * VECSIZE;

	// Allocate space for device copies of a, b, c
	hipMalloc((void **) &gpu_vector1, size);
	hipMalloc((void **) &gpu_vector2, size);
	hipMalloc((void **) &gpu_vector_out, size);

	// Populate the input vectors with random integers
	random_ints(vector1, VECSIZE);
	random_ints(vector2, VECSIZE);

	// Copy inputs to device
	hipMemcpy(gpu_vector1, &vector1, size, hipMemcpyHostToDevice);
	hipMemcpy(gpu_vector2, &vector2, size, hipMemcpyHostToDevice);

	vector_add<<<BLOCKS, THREADSPERBLOCK>>> (gpu_vector1, gpu_vector2, gpu_vector_out);

	// Copy result back to host
	hipMemcpy(&vector_out, gpu_vector_out, size, hipMemcpyDeviceToHost);

	// Cleanup
	hipFree(gpu_vector1);
	hipFree(gpu_vector2);
	hipFree(gpu_vector_out);

	show_vector(vector1, VECSIZE);
	show_vector(vector2, VECSIZE);
	printf("\n");
	show_vector(vector_out, VECSIZE);

	printf("\n");

	return 0;
}
