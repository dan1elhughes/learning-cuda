
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void vector_add(int *a, int *b, int *c) {
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void random_ints(int* a, int n) {
	int i;
	for (i = 0; i < n; ++i) {
		a[i] = rand();
	}
}

int getRand() {
	// return rand() % 50;

	// Chosen by a fair dice roll; guaranteed to be random.
	return 1;
}

int main(void) {
	int vector1[8];
	int vector2[8];
	int vector_out[8];

	int *gpu_vector1, *gpu_vector2, *gpu_vector_out;

	int size = sizeof(int) * 8;

	// Allocate space for device copies of a, b, c
	 hipMalloc((void **) &gpu_vector1, size);
	 hipMalloc((void **) &gpu_vector2, size);
	 hipMalloc((void **) &gpu_vector_out, size);

	 // Populate the input vectors with random integers
	 random_ints(vector1, 8);
	 random_ints(vector2, 8);

	 for(int i = 0; i < 8; i++) {
		 printf("%d\t", vector1[i]);
	 }
	 printf("\n");

	 for(int i = 0; i < 8; i++) {
		 printf("%d\t", vector2[i]);
	 }
	 printf("\n\n");

	// Copy inputs to device
	 hipMemcpy(gpu_vector1, &vector1, size, hipMemcpyHostToDevice);
	 hipMemcpy(gpu_vector2, &vector2, size, hipMemcpyHostToDevice);

	// Launch vector_add() kernel on GPU
	 vector_add <<<8, 1>>> (gpu_vector1, gpu_vector2, gpu_vector_out);

	// Copy result back to host
	 hipMemcpy(&vector_out, gpu_vector_out, size, hipMemcpyDeviceToHost);

	// Cleanup
	 hipFree(gpu_vector1);
	 hipFree(gpu_vector2);
	 hipFree(gpu_vector_out);

	for(int i = 0; i < 8; i++) {
		printf("%d\t", vector_out[i]);
	 }
	 printf("\n");

	 return 0;
}
